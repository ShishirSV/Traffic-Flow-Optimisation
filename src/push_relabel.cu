#include "/home/shishir/Random/CUDA/Major Project 2/code_par.cuh"

void push_relabel(int V, int source, int sink, int *cpu_height, int *cpu_excess_flow, int *cpu_adjmtx, int *cpu_rflowmtx, int *Excess_total, int *gpu_height, int *gpu_excess_flow, int *gpu_adjmtx, int *gpu_rflowmtx)
{
    /* Instead of checking for overflowing vertices(as in the sequential push relabel),
     * sum of excess flow values of sink and source are compared against Excess_total 
     * If the sum is lesser than Excess_total, 
     * it means that there is atleast one more vertex with excess flow > 0, apart from source and sink
     */

    /* declaring the mark and scan boolean arrays used in the global_relabel routine outside the while loop 
     * This is not to lose the mark values if it goes out of scope and gets redeclared in the next iteration 
     */

    bool *mark,*scanned;
    mark = (bool*)malloc(V*sizeof(bool));
    scanned = (bool*)malloc(V*sizeof(bool));

    // initialising mark values to false for all nodes
    for(int i = 0; i < V; i++)
    {
        mark[i] = false;
    }

    while((cpu_excess_flow[source] + cpu_excess_flow[sink]) < *Excess_total)
    {
        // copying height values to CUDA device global memory
        hipMemcpy(gpu_height,cpu_height,V*sizeof(int),hipMemcpyHostToDevice);

        printf("Invoking kernel\n");

        // invoking the push_relabel_kernel
        push_relabel_kernel<<<number_of_blocks_nodes,threads_per_block>>>(V,source,sink,gpu_height,gpu_excess_flow,gpu_adjmtx,gpu_rflowmtx);

        hipDeviceSynchronize();


        // copying height, excess flow and residual flow values from device to host memory
        hipMemcpy(cpu_height,gpu_height,V*sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(cpu_excess_flow,gpu_excess_flow,V*sizeof(int),hipMemcpyDeviceToHost);
        hipMemcpy(cpu_rflowmtx,gpu_rflowmtx,V*V*sizeof(int),hipMemcpyDeviceToHost);

        printf("After invoking\n");
        //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        printf("Excess total : %d\n",*Excess_total);
        // perform the global_relabel routine on host
        global_relabel(V,source,sink,cpu_height,cpu_excess_flow,cpu_adjmtx,cpu_rflowmtx,Excess_total,mark,scanned);

        printf("\nAfter global relabel\n");
        //print(V,cpu_height,cpu_excess_flow,cpu_rflowmtx,cpu_adjmtx);
        printf("Excess total : %d\n",*Excess_total);
    }

}